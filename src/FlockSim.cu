#include "hip/hip_runtime.h"
#include"FlockSim.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<iomanip>
#include<time.h>

//#define NDEBUG
using namespace std;

FlockSim::FlockSim(){
  

}


FlockSim::FlockSim(int size,float wall_size)
{
  wallx = wall_size;
  wally = wall_size;    
  initialFlock(size);
  hipMalloc((void**)&dev_flock,F.size*sizeof(Agent));
  hipMemcpy(dev_flock, F.flock, F.size*sizeof(Agent),hipMemcpyHostToDevice);  
}
__device__ float check_angle (float ang)
{
  if (ang >= (float)360)
    return ang - (float) 360;
  if (ang < 0)
    return ang + (float) 360;
  else
    return ang;
}

__global__ void update_flock_gpu (Agent* F, float wallx,float wally,int size,float dt)
{
  int num = threadIdx.x + blockDim.x * blockIdx.y;
  if (num < size)
  {
    F[num].x += cos(F[num].v)*dt;
    F[num].y += sin(F[num].v)*dt;
    if (F[num].x >= wallx || F[num].x <= (float)0.0)
      F[num].angle = ((float)180.0 - F[num].angle);
    if (F[num].y >= wally || F[num].y <= (float)0.0)
      F[num].angle =  (-(float)1.0* F[num].angle);
    check_angle(F[num].angle);
  }
}



void FlockSim::update_flock(float dt)
{
  update_flock_gpu<<<cusp.Grid,cusp.Block>>>(dev_flock,wallx,wally,F.size,dt);
}

  
void FlockSim::initialFlock(int size)
{
  /*
    space 0~wallx 0~wally 0~wallz
  */
  F.size = size;
  F.flock =(Agent*) malloc(size*sizeof(Agent));
  srand((unsigned)time(0));
  for (int i = 0; i < F.size; ++i)
  {
    F.flock[i].angle = (float)rand()/(float)RAND_MAX*360.0;
    F.flock[i].x = (float)rand()/(float)RAND_MAX*wallx;
    F.flock[i].y = (float)rand()/(float)RAND_MAX*wally;
    F.flock[i].v = (float)rand()/(float)RAND_MAX; // 0~1
  }
}

void FlockSim::printFlock()
{
  hipMemcpy(F.flock,dev_flock,F.size*sizeof(Agent),hipMemcpyDeviceToHost);  
//  cout  <<setw(8)<< "n"\
        <<setw(8) << "ang"\
        <<setw(8) << "x"\
        <<setw(8) << "y"\
        <<setw(8) << "v"\    
//        << endl;
//  for (int i = 0; i < F.size; ++i)
//  {
//    cout << setw(8) << i\
         <<setw(8) <<(int) F.flock[i].angle             \
         <<setw(8) << setprecision(2)<< F.flock[i].x    \
         <<setw(8) << F.flock[i].y                      \
         <<setw(8) << F.flock[i].v                      \      
 //        <<endl;
 // }
//  cout << endl;
}


