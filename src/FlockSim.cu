#include"FlockSim.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<iomanip>
#include<time.h>

//#define NDEBUG
using namespace std;

void initialFlock(Flock &F,int size,float wallsize)
{
  /*
    space 0~wallx 0~wally 0~wallz
  */
  F.size = size;
  F.wallx = wallsize;
  F.wally = wallsize;
  F.flock =(Agent*) malloc(size*sizeof(Agent));
  srand((unsigned)time(0));
  for (int i = 0; i < F.size; ++i)
  {
    F.flock[i].angle = (float)rand()/(float)RAND_MAX*360.0;
    F.flock[i].x = (float)rand()/(float)RAND_MAX*F.wallx;
    F.flock[i].y = (float)rand()/(float)RAND_MAX*F.wally;        
  }
}

void printFlock(Flock &F)
{
  cout  <<setw(8)<< "n"\
        <<setw(8) << "ang"\
        <<setw(8) << "x"\
        <<setw(8) << "y"\
        << endl;
  for (int i = 0; i < F.size; ++i)
  {
    cout << setw(8) << i\
         <<setw(8) <<(int) F.flock[i].angle             \
         <<setw(8) << setprecision(2)<< F.flock[i].x    \
         <<setw(8) << F.flock[i].y                      \
         <<endl;
  }
}


