#include "hip/hip_runtime.h"
#include"FlockSim.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<iomanip>
#include<time.h>

//#define NDEBUG
using namespace std;

FlockSim::FlockSim(int size,float wall_size)
{
  wallx = wall_size;
  wally = wall_size;    
  initialFlock(size);
}

__global__ void update_flock ()
{
}

void FlockSim::initialFlock(int size)
{
  /*
    space 0~wallx 0~wally 0~wallz
  */
  F.size = size;
  F.flock =(Agent*) malloc(size*sizeof(Agent));
  srand((unsigned)time(0));
  for (int i = 0; i < F.size; ++i)
  {
    F.flock[i].angle = (float)rand()/(float)RAND_MAX*360.0;
    F.flock[i].x = (float)rand()/(float)RAND_MAX*wallx;
    F.flock[i].y = (float)rand()/(float)RAND_MAX*wally;        
  }
}

void FlockSim::printFlock()
{
  cout  <<setw(8)<< "n"\
        <<setw(8) << "ang"\
        <<setw(8) << "x"\
        <<setw(8) << "y"\
        << endl;
  for (int i = 0; i < F.size; ++i)
  {
    cout << setw(8) << i\
         <<setw(8) <<(int) F.flock[i].angle             \
         <<setw(8) << setprecision(2)<< F.flock[i].x    \
         <<setw(8) << F.flock[i].y                      \
         <<endl;
  }
}


