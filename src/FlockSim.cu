#include "hip/hip_runtime.h"
#include"FlockSim.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include<iomanip>
#include<time.h>

//#define NDEBUG
using namespace std;


FlockSim::FlockSim(int size,float wall_size)
{
  wallx = wall_size;
  wally = wall_size;    
  initialFlock(size);
  hipMalloc((void**)&dev_flock,F.size*sizeof(Agent));
  hipMemcpy(dev_flock, F.flock, F.size*sizeof(Agent),hipMemcpyHostToDevice);  
}
__device__ float check_angle (float ang)
{
  if (ang >= (float)360)
    return ang - (float) 360;
  if (ang < 0)
    return ang + (float) 360;
  else
    return ang;
}

__global__ void update_flock_gpu (Agent* F, float wallx,float wally,int size,float dt)
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    F[num].x += cos(F[num].angle)*F[num].v*dt;
    F[num].y += sin(F[num].angle)*F[num].v*dt;
    if (F[num].x >= wallx || F[num].x <= (float)0.0)
      F[num].angle = ((float)180.0 - F[num].angle);
    if (F[num].y >= wally || F[num].y <= (float)0.0)
      F[num].angle =  (-(float)1.0* F[num].angle);
    check_angle(F[num].angle);
  }
}



void FlockSim::update_flock(float dt)
{
  dim3 Grid(cusp.Grid_Dim_x, cusp.Grid_Dim_y);		//Grid structure
  dim3 Block(cusp.Block_Dim_x,cusp.Block_Dim_y);	//Block structure, threads/block limited by specific device        
  update_flock_gpu<<<Grid,Block>>>(dev_flock,wallx,wally,F.size,dt);
}

  
void FlockSim::initialFlock(int size)
{
  /*
    space 0~wallx 0~wally 0~wallz
  */
  F.size = size;
  F.flock =(Agent*) malloc(size*sizeof(Agent));
  srand((unsigned)time(0));
  for (int i = 0; i < F.size; ++i)
  {
    F.flock[i].angle = (float)rand()/(float)RAND_MAX*360.0;
    F.flock[i].x = (float)rand()/(float)RAND_MAX*wallx;
    F.flock[i].y = (float)rand()/(float)RAND_MAX*wally;
    F.flock[i].v = (float)rand()/(float)RAND_MAX; // 0~1
  }
}

void FlockSim::printFlock()
{
  hipMemcpy(F.flock,dev_flock,F.size*sizeof(Agent),hipMemcpyDeviceToHost);  
  cout  <<setw(8)<< "n"\
        <<setw(8) << "ang"\
        <<setw(8) << "x"\
        <<setw(8) << "y"\
        <<setw(8) << "v"\    
        << endl;
  for (int i = 0; i < F.size; ++i)
  {
    cout << setw(8) << i\
         <<setw(8) <<(int) F.flock[i].angle             \
         <<setw(8) << setprecision(2)<< F.flock[i].x    \
         <<setw(8) << F.flock[i].y                      \
         <<setw(8) << F.flock[i].v                      \      
         <<endl;
  }
  cout << endl;
}

void FlockSim::copy2host()
{
  hipMemcpy(F.flock,dev_flock,F.size*sizeof(Agent),hipMemcpyDeviceToHost);  
}


