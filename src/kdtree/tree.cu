#include "hip/hip_runtime.h"
#include "tree.h"
//#ifndef NDEBUG
#include <stdio.h>
#include <unistd.h>
#include <stdio.h>

WorldGeo::WorldGeo(int dim):_dim(dim)
{
  assert(_dim == 2 || _dim == 3);
  _wall = new float[2*_dim];
}

WorldGeo::~WorldGeo()
{
  //delete[] _wall;
}
int WorldGeo::getDim(){return _dim;}
float WorldGeo::getWall(int dim,int m){return _wall[dim*2+m];}
void WorldGeo::setWall(float* wall)
{
  for (int i = 0; i < 2*_dim; ++i)
    _wall[i] = wall[i];
}

KdTree::KdTree(int thread_n, int size, WorldGeo* wg):_thread_n(thread_n),_size(size),_wg(wg)
{
  /* pthread handles */
  _nodes = new Node[_size];
  _dim = wg->getDim();
  for (int i = 0; i < _size; ++i)
    _nodes[i].init(_dim,i,_size);
  assert(_size != 0);
  assert(_dim == 2 || _dim == 3);  
}

KdTree::~KdTree()
{
  delete[] _nodes;
  _nodes = NULL;
}

void KdTree::findRoot()
{
  _root = _nodes[0].median(SAMPLESIZE,NULL,false);
  _nodes[_root].buildRootList(_size);
  _unfinish.push(&(_nodes[_root]));
}

void* KdTree::construct_thread(Node* job,struct drand48_data* buffer)
{
  queue<Node*> unfinish;
  unfinish.push(job);
  assert(unfinish.size() == 1);
  Node* cur;
  Node* left,* right;
  while(unfinish.size() > 0)
  {
    cur = unfinish.front();
    unfinish.pop();
    cur->separateList();
    if (cur->getLList()->size() != 0)
      left = &(_nodes[cur->leftmedian(buffer)]);
    else
      left = NULL;
    
    if (cur->getRList()->size() != 0)
      right = &(_nodes[cur->rightmedian(buffer)]);
    else
      right = NULL;
    cur->setChild(left,right);
    if (left != NULL)
    {
      unfinish.push(left);
    }
    if (right != NULL)
    {
      unfinish.push(right);
    }
  };
  return NULL;
}

void KdTree::construct()
{
  Node* cur;
  Node* left,*right;
  
  while(_unfinish.size() < _thread_n)
  {
    cur = _unfinish.front();
    _unfinish.pop();
    cur->separateList();
    if (cur->getLList()->size() != 0)
      left = &(_nodes[cur->leftmedian()]);
    else
      left = NULL;
    if (cur->getRList()->size() != 0)
      right = &(_nodes[cur->rightmedian()]);
    else
      right = NULL;

    cur->setChild(left,right);
    
    if (left != NULL)
      _unfinish.push(left);
    if (right != NULL)    
      _unfinish.push(right);    
  }
  assert(_unfinish.size() == _thread_n);
}

void KdTree::printNodes()
{
  for (int i = 0; i < _size; ++i)
    cout << _nodes[i] << endl;
}

void KdTree::testInit()
{
  
}

void KdTree::randInit()
{
  srand (time(NULL));
  // position
  for (int i = 0; i < _size; ++i)
    for (int j = 0; j < _dim; ++j)
      _nodes[i].setPos(j,randRange(_wg->getWall(j,0),_wg->getWall(j,1)));
  for (int i = 0; i < _size; ++i)
    for (int j = 0; j < _dim; ++j)
      _nodes[i].setDir(j,randRange(-2.0,2.0));  
}

int KdTree::getRoot(){return _root;}
Node* KdTree::getJob(){
  Node* tmp;
  tmp = _unfinish.front();
  _unfinish.pop();
  return tmp;
}


bool KdTree::checkTree()
{
  // stupid kd tree check.
  int check;
  int cur,tmp;
  int ax;
  int lr;                       // 0 if left.1 if right
  assert(_dim == 2 || _dim == 3);
  for (int i = 0; i < _size; ++i)
  {
    check = i;
    cur = i;
    while (cur != _root)
    {
      tmp = _nodes[cur].getParent();
      if (cur == _nodes[tmp].getLChild())
        lr = 0;
      if (cur == _nodes[tmp].getRChild())
        lr = 1;
      cur = tmp;
      assert(check>= 0 && check < _size);
      assert(cur>= 0 && cur < _size);                        
      ax = Node::getDepth(cur)%_dim;
      if (lr == 0)
      {
        assert(check>= 0 && check < _size);
        assert(cur>= 0 && cur < _size);              
        if (Node::getPos(check,ax) > Node::getPos(cur,ax))
        {
          cout << "check:" << check << " wrong at:" << cur << endl;
          return false;
        }
      }
      else
      {
        assert(check>= 0 && check < _size);
        assert(cur>= 0 && cur < _size);                            
        if (Node::getPos(check,ax) < Node::getPos(cur,ax))
        {
          cout << "check:" << check << " wrong at:" << cur << endl;          
          return false;
        }
      }
    };
  }
  return true;
}

void KdTree::findWithin_slow(int d,float dis)
{
  int count = 0;
  cout << "slow version"<<endl;
  for (int i = 0; i < _size; ++i)
  {
    if (i != d)
    {
      if (_nodes[d].distance(i) < dis)
      {
        //cout << i << endl;
        count++;
      }
    }
  }
  cout << count<<endl;
}

int KdTree::goDown(int& cur,int& d,float& dis)
{
  int ax;
  int tmp;
  int count = 0;
  if (cur != d && _nodes[d].distance(cur) < dis)
  {
    //cout << cur << endl;
    count++;
  }  
  while (!_nodes[cur].isEnd())
  {
    ax = Node::getDepth(cur)%_dim;
    assert(d>= 0 && d < _size);
    assert(cur>= 0 && cur < _size);                    
    if (Node::getPos(d,ax) > Node::getPos(cur,ax))
    {
      tmp = _nodes[cur].getRChild();
      if (tmp == cur)
        cur = _nodes[cur].getLChild();
      else cur = tmp;
    }
    else
    {
      tmp = _nodes[cur].getLChild();
      if (tmp == cur)
        cur = _nodes[cur].getRChild();
      else cur = tmp;          
    }
    if (cur != d && _nodes[d].distance(cur) < dis)
    {
      //cout << cur << endl;
      count++;
    }
  }
  return count;
}

bool KdTree::move(int& cur , int& d,float& dis)
{
  assert(d>= 0 && d < _size);                
  int parent = _nodes[cur].getParent();
  int ax = Node::getDepth(parent)%_dim;
  float d_ax = Node::getPos(d,ax);
  assert(parent>= 0 && parent < _size);                  
  float curp_ax = Node::getPos(parent,ax);

  if (fabs(d_ax - curp_ax) <= dis)
  {
    int rc = _nodes[parent].getRChild();
    int lc = _nodes[parent].getLChild();              
    if (d_ax > curp_ax)
    {
      if (cur == rc && parent != lc)
      {
        cur = lc;
        return true;
      }
      else
      {
        cur = parent;
        return false;
      }
    }
    else
    {
      if (cur == lc && parent != rc)
      {
        cur = rc;
        return true;
      }
      else
      {
        cur = parent;
        return false;
      }
    }
  }
  else
  {
    cur= parent;
    return false;
  }
}

void KdTree::findWithin(int d,float dis)
{
  int count = 0;
  int cur = _root;
  count += goDown(cur,d,dis);
  while (cur != _root)
  {
    if (move(cur,d,dis))
    {
      count += goDown(cur,d,dis);
    }
  }
  cout << count << endl;
}

int KdTree::deepest()
{
  int d = 0;
  int tmp;
  for (int i = 0; i < _size; ++i)
  {
    tmp = Node::getDepth(i);
    if (tmp > d)
      d = tmp;
  }
  return d;
}
void KdTree::depthArray(int* arr)
{
  for (int i = 0; i < _size; ++i)
    arr[i] = Node::getDepth(i);
}

void KdTree::clearTree()
{
  for (int i = 0; i < _size; ++i)
    _nodes[i].clear();
}

void* launchThread(void* arg)
{
  // initialize drand
  ThreadArgs * myarg = (ThreadArgs*)arg;
  struct timeval tv;
  gettimeofday(&tv, NULL);
  struct drand48_data drand_buffer;  
  srand48_r(tv.tv_sec * myarg->rank + tv.tv_usec, &drand_buffer);
  cout << "        thread" <<myarg->rank<<endl;
  myarg->myTree->construct_thread(myarg->job, &drand_buffer);
  cout << "        return" << myarg->rank<<endl;
  return NULL;
}

void ConstructTree(int thread_n , KdTree* myTree, pthread_t* thread_handles)
{
  ThreadArgs* args = new ThreadArgs[thread_n];
  for (int i = 0; i < thread_n; ++i)
  {
    args[i].rank = i;
    args[i].job = myTree->getJob();
    //    assert(args[i].job->getIdx() >= 0&&args[i].job->getIdx() <10000);
    args[i].myTree = myTree;
  }
  int thread;
  void                 *status[thread_n];
  //    cerr <<"      start thread"<<endl;
  for (thread = 0; thread < thread_n; thread++)
  {
    pthread_create(&thread_handles[thread],NULL, launchThread,(void*) &args[thread]);
  }
  
  //    cerr <<"      join thread"<<endl;
  for (thread = 0; thread < thread_n; thread++)
  {
    pthread_join(thread_handles[thread],&status[thread]);
  }
  for (int i=0; i <thread_n; ++i) {
    if (status[i] != NULL) {
      printf("Unexpected thread status\n");
    }
  }  
  //  cerr << "       delete"<<endl;
  delete [] args;
  args = NULL;
}



