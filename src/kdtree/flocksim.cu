#include "hip/hip_runtime.h"
#include "flocksim.h"

FlockSim::FlockSim(int size, int thread_n,WorldGeo& wg,Para para):_size(size),_thread_n(thread_n),_wg(wg)
{
  _thread_handles = (pthread_t*)malloc(_thread_n*sizeof(pthread_t));
  cout << "Using " << _thread_n << " threads!" << endl;
  _kt = new KdTree(_thread_n,size,&_wg);
  cout << "random initialize"<<endl;
  _kt->randInit();
  cout << "allocate memory on GPU"<< endl;
  _psize = Node::getPSize();
  _pos = Node::getPos();
  _xyz_dir = Node::getDir();  
  _tree = Node::getTree();
  _depth = new int[_size];
  hipMalloc((void**)&_dev_pos,_size*_psize*sizeof(float));
  hipMalloc((void**)&_dev_tree,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_depth,_size*sizeof(int));  
  hipMalloc((void**)&_dev_xyz_dir,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_ang_dir,_size*3*sizeof(int));  
  _ang_dir = new float[_size*3*sizeof(float)];
  // cuda grid sructure
  Block_Dim_x = 512;
  Block_Dim_y = 1;  
  Grid_Dim_x = (int)_size/Block_Dim_x +1;
  if (Grid_Dim_x > 65565)
    cerr << "too many block!" << endl;
  Grid_Dim_y = 1;
  _para = para;
}

FlockSim::~FlockSim()
{
  hipFree(&_dev_pos);         // will render need this mem?
  hipFree(&_dev_tree);
  hipFree(&_dev_xyz_dir);
  hipFree(&_dev_ang_dir);  
  delete [] _ang_dir;
}

void FlockSim::initializeGpuData()
{
  hipMemcpy(_dev_pos, _pos, _size*_psize*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(_dev_xyz_dir, _xyz_dir, _size*3*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("para"), &_para, sizeof(Para), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("pos"), &_dev_pos, sizeof(float*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("xyz_dir"), &_dev_xyz_dir, sizeof(float*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("ang_dir"), &_dev_ang_dir, sizeof(float*), size_t(0),hipMemcpyHostToDevice);    
  hipMemcpyToSymbol(HIP_SYMBOL("tree"), &_dev_tree, sizeof(int*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("depth"), &_dev_depth, sizeof(int*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("size"), &_size, sizeof(int), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("psize"), &_psize, sizeof(int), size_t(0),hipMemcpyHostToDevice);    
}

void FlockSim::cpytree2dev()
{
  hipMemcpy(_dev_tree, _tree, _size*3*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(_dev_depth, _depth, _size*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("root"), &_root, sizeof(int), size_t(0),hipMemcpyHostToDevice);  
}

void FlockSim::cpy2host()
{
  hipMemcpy(_pos, _dev_pos, _size*_psize*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(_ang_dir, _dev_ang_dir, _size*3*sizeof(float),hipMemcpyDeviceToHost);  
}

void FlockSim::depthArray()
{
  _kt->depthArray(_depth);
}

void FlockSim::makeTree()
{
  _kt->findRoot();  
  _kt->construct();
  ConstructTree(_thread_n,_kt,_thread_handles);
  _kt->printNodes();
  // if(_kt->checkTree())
  //   cout << "correct" << endl;
  _root = _kt->getRoot();
  depthArray();
}

__constant__ Para para;
__constant__ float* pos;
__constant__ float* xyz_dir;
__constant__ float* ang_dir;
__constant__ int* tree;
__constant__ int* depth;
__constant__ int size;
__constant__ int psize;
__constant__ int root;          // need to update



__global__  void convertDir()
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    float r = 0.0;
    for (int i = 0; i < 3; ++i)
    {
      r += xyz_dir[num*3+i]*xyz_dir[num*3+i];
    }
    r = sqrt(r);
    ang_dir[num*3] = -180.0*acos(xyz_dir[num*3]/r)/M_PI;    
    ang_dir[num*3+1] = xyz_dir[num*3+2];
    ang_dir[num*3+2] = -1.0*xyz_dir[num*3+1];    
  }
}


__device__ float3 operator+(const float3 &a, const float3 &b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ float3 getPos (int &a)
{
  return make_float3(pos[a*psize],pos[a*psize+1],pos[a*psize+2]);
}

__device__ void setPos (int &a,float3 &p)
{
  pos[a*psize] = p.x;
  pos[a*psize+1] = p.y;
  pos[a*psize+2] = p.z;  
}


__device__ float3 getDIr (int &a)
{
  return make_float3(xyz_dir[a*3],xyz_dir[a*3+1],xyz_dir[a*3+2]);
}

__device__ void setDIr (int &a,float3 &p)
{
  xyz_dir[a*3] = p.x;
  xyz_dir[a*3+1] = p.y;
  xyz_dir[a*3+2] = p.z;  
}


__device__ float distance(int &a, int &b)
{
  float3 tmp = getPos(a)-getPos(b);
  return sqrt(tmp.x*tmp.x+tmp.y*tmp.y+tmp.z*tmp.z);
}

__device__ void calculateAvg(int num,Avg &avg)
{
  int cur = root;
  
  // int cur = root;
  // cur = goDown(cur,num,dis);n
  // while (cur != root)
  // {
  //   if (move(cur,num,dis))
  //     cur = goDown(cur,num,dis);
  // }
}

__global__ void flockUpdate()
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    Avg avg;
    avg.Rpos = make_float3(0,0,0);
    avg.rpos = make_float3(0,0,0);
    avg.Rvel = make_float3(0,0,0);
    avg.rvel = make_float3(0,0,0);
    avg.count = 0;
    calculateAvg(num,avg);
    // use para variable like para.R para.r
    if (num == 0)
    {
      printf("R:%f\n",para.R);
      printf("r:%f\n",para.r);
      printf("root:%d\n",root);
      printf("psize:%d\n",psize);
    }
    
  }
}

void FlockSim::update()
{
    dim3 Grid(Grid_Dim_x, Grid_Dim_y);		//Grid structure
    dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device
    flockUpdate<<<Grid,Block>>>();
    convertDir<<<Grid,Block>>>(); // convert xyz velocity to angle
}
