#include "hip/hip_runtime.h"
#include "flocksim.h"

FlockSim::FlockSim(int size, int thread_n,WorldGeo& wg,Para para):_size(size),_thread_n(thread_n),_wg(wg)
{
  _thread_handles = (pthread_t*)malloc(_thread_n*sizeof(pthread_t));
  cout << "Using " << _thread_n << " threads!" << endl;
  _kt = new KdTree(_thread_n,size,&_wg);
  cout << "random initialize"<<endl;
  _kt->randInit();
  cout << "allocate memory on GPU"<< endl;
  _psize = Node::getPSize();
  _pos = Node::getPos();
  _xyz_dir = Node::getDir();  
  _tree = Node::getTree();
  _depth = new int[_size];
  hipMalloc((void**)&_dev_pos,_size*_psize*sizeof(float));
  hipMalloc((void**)&_dev_tree,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_depth,_size*sizeof(int));  
  hipMalloc((void**)&_dev_xyz_dir,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_ang_dir,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_wall,6*sizeof(float));
  hipMalloc((void**)&_dev_isend,_size*sizeof(int));    
  _ang_dir = new float[_size*3*sizeof(float)];

  // cuda grid sructure
  Block_Dim_x = 128;
  Block_Dim_y = 1;  
  Grid_Dim_x = (int)_size/Block_Dim_x +1;
  if (Grid_Dim_x > 65565)
    cerr << "too many block!" << endl;
  Grid_Dim_y = 1;
  _para = para;
}

FlockSim::~FlockSim()
{
  hipFree(&_dev_pos);         // will render need this mem?
  hipFree(&_dev_tree);
  hipFree(&_dev_isend);  
  hipFree(&_dev_xyz_dir);
  hipFree(&_dev_ang_dir);
  hipFree(&_dev_depth);
  hipFree(&_dev_wall);
  delete [] _ang_dir;
  delete [] _depth;
}

void FlockSim::initializeGpuData()
{
  hipMemcpy(_dev_pos, _pos, _size*_psize*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(_dev_xyz_dir, _xyz_dir, _size*3*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(_dev_wall, _wg._wall , 6*sizeof(float),hipMemcpyHostToDevice);  
  hipMemcpyToSymbol(HIP_SYMBOL("para"), &_para, sizeof(Para), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("pos"), &_dev_pos, sizeof(float*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("xyz_dir"), &_dev_xyz_dir, sizeof(float*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("ang_dir"), &_dev_ang_dir, sizeof(float*), size_t(0),hipMemcpyHostToDevice);    
  hipMemcpyToSymbol(HIP_SYMBOL("tree"), &_dev_tree, sizeof(int*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("depth"), &_dev_depth, sizeof(int*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("size"), &_size, sizeof(int), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("psize"), &_psize, sizeof(int), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("wall"), &(_dev_wall), sizeof(float*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("isend"), &_dev_isend, sizeof(int*), size_t(0),hipMemcpyHostToDevice);  
}

void FlockSim::cpytree2dev()
{
  hipMemcpy(_dev_tree, _tree, _size*3*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(_dev_depth, _depth, _size*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("root"), &_root, sizeof(int), size_t(0),hipMemcpyHostToDevice);  
}

void FlockSim::cpy2host()
{
  hipMemcpy(_pos, _dev_pos, _size*_psize*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(_ang_dir, _dev_ang_dir, _size*3*sizeof(float),hipMemcpyDeviceToHost);  
}

void FlockSim::depthArray()
{
  _kt->depthArray(_depth);
}

void FlockSim::makeTree()
{
  _kt->findRoot();  
  _kt->construct();
  ConstructTree(_thread_n,_kt,_thread_handles);
  //_kt->printNodes();
  // if(_kt->checkTree())
  //   cout << "correct" << endl;
  _root = _kt->getRoot();
}

__constant__ Para para;
__constant__ float* pos;
__constant__ float* xyz_dir;
__constant__ float* ang_dir;
__constant__ int* tree;
__constant__ int* depth;
__constant__ int size;
__constant__ int psize;
__constant__ float* wall;
__constant__ int root;          // need to update
__constant__ int* isend;


__global__  void convertDir()
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    float r = 0.0;
    for (int i = 0; i < 3; ++i)
    {
      r += xyz_dir[num*3+i]*xyz_dir[num*3+i];
    }
    r = sqrt(r);
    ang_dir[num*3] = -180.0*acos(xyz_dir[num*3]/r)/M_PI;    
    ang_dir[num*3+1] = xyz_dir[num*3+2];
    ang_dir[num*3+2] = -1.0*xyz_dir[num*3+1];    
  }
}


__device__ float3 operator+(const float3 &a, const float3 &b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ float3 operator/(const float3 &a, const float &b) {
   if(b!=0){
      return make_float3(a.x/b, a.y/b, a.z/b);
   }
   else{
      return make_float3(0,0,0);
   }
}

__device__ float3 operator*(const float3 &a, const float &b) {
  return make_float3(a.x*b, a.y*b, a.z*b);
}

inline __device__ float3 getPos (int &a)
{
  return make_float3(pos[a*psize],pos[a*psize+1],pos[a*psize+2]);
}

inline __device__ float getPosAx (int &a, int &ax){  return pos[a*psize+ax];}

inline __device__ void setPos (int &a,float3 p)
{
  pos[a*psize] = p.x;
  pos[a*psize+1] = p.y;
  pos[a*psize+2] = p.z;  
}


inline __device__ float3 getDir (int &a)
{
  return make_float3(xyz_dir[a*3],xyz_dir[a*3+1],xyz_dir[a*3+2]);
}

inline __device__ void setDir (int &a,float3 &p)
{
  xyz_dir[a*3] = p.x;
  xyz_dir[a*3+1] = p.y;
  xyz_dir[a*3+2] = p.z;  
}

inline __device__ int getLChild(int &num){  return tree[num*3+1];}
inline __device__ int getRChild(int &num){  return tree[num*3+2];}
inline __device__ int getParent(int &num){return tree[num*3];}
inline __device__ int getDepth(int &num){  return depth[num];}
inline __device__ bool isEnd(int &num)
{
  if (isend[num] == 1) return true;
  else if (isend[num] == 0) return false;
  else if (num==getLChild(num) && num==getRChild(num))
  {
    isend[num] = 1;
    return true;
  }
  else
  {
    isend[num] = 0;
    return false;
  }
}

inline __host__ __device__ float dot(float3 a, float3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __device__ float dis(int &a, int &b)
{
  float3 tmp = getPos(a)- getPos(b);
  return sqrtf(dot(tmp, tmp));  
}

__device__ void goDown(int &cur,int& num,Avg& avg)
{
  int ax,tmp;
  float dist;
  dist = dis(cur,num);
  if (cur != num &&  dist< para.R)
  {
    avg.countR++;
    avg.Rpos = avg.Rpos + getPos(cur);
    avg.Rvel = avg.Rvel + getDir(cur);    
    if (dist < para.r)
    {
      avg.countr++;
      avg.rpos = avg.rpos +(getPos(num)-getPos(cur))/(dist/10);
      //avg.rvel = avg.rvel + getDir(num);
    }
  }
  int i=0;
  while(!isEnd(cur)&&i<1)
  {
    ax = getDepth(cur)%3;
    if (getPosAx(num,ax)>getPosAx(cur,ax))
    {
      tmp = getRChild(cur);
     if (tmp == cur)
        cur = getLChild(cur);
      else cur = tmp;
    }
    else
    {
      tmp = getLChild(cur);
      if (tmp == cur)
        cur = getRChild(cur);
      else cur = tmp;      
    }
    dist = dis(cur,num);    
    if (cur != num &&  dist< para.R)
    {
      avg.countR++;
      avg.Rpos = avg.Rpos + getPos(cur);
      avg.Rvel = avg.Rvel + getDir(cur);    
      if (dist < para.r)
      {
        avg.countr++;
        avg.rpos = avg.rpos +(getPos(num)-getPos(cur))/(dist/10);
     //   avg.rpos = avg.rpos + getPos(num)/dist;
      //  avg.rvel = avg.rvel + getDir(num);
      }
    }
    //i++;
  }
}
__device__ bool move(int &cur,int &num)
{
  int parent = getParent(cur);
  int ax = getDepth(parent)%3;
  float d_ax = getPosAx(num,ax);
  float curp_ax = getPosAx(parent,ax);
  if (fabs(d_ax - curp_ax) <= para.R)
  {
    int rc = getRChild(parent);
    int lc = getLChild(parent);              
    if (d_ax > curp_ax)
    {
      if (cur == rc && parent != lc)
      {
        cur = lc;
        return true;
      }
      else
      {
        cur = parent;
        return false;
      }
    }
    else
    {
      if (cur == lc && parent != rc)
      {
        cur = rc;
        return true;
      }
      else
      {
        cur = parent;
        return false;
      }
    }
  }
  else
  {
    cur= parent;
    return false;
  }
}

// normalize
inline __host__ __device__ float3 normalize(float3 v)
{
    float invLen = 1.0f / sqrtf(dot(v, v));
    return v * invLen;
}
// __device__ float3 normalize(float3 a)
// {
//   float tx,ty,tz;
//   float t;
//   t = sqrt(a.x*a.x+a.y*a.y+a.z*a.z);
//   if(t!=0){
//   tx = a.x/t;
//   ty = a.y/t;
//   tz = a.z/t;
//   }
//   else{
//   tx=0,ty=0,tz=0;
//   }
//   return make_float3(tx,ty,tz);
// }

__device__ void calculateAvg(int num,Avg &avg)
{
  int cur = root;
  goDown(cur,num,avg);
  while (cur != root)
  {
    if (move(cur,num))
      goDown(cur,num,avg);
  }
}
__device__ void wallCheck(int& num,float3 &pos,float3 &dir)
{
  if(pos.x < wall[0] || pos.x > wall[1])
    dir.x = dir.x*-1.0;
  if(pos.y < wall[2] || pos.y > wall[3])
    dir.y = dir.y*-1.0;
  if(pos.z < wall[4] || pos.z > wall[5])
    dir.z = dir.z*-1.0;  
  
  if(pos.x<wall[0])
    pos.x=wall[0];
  if(pos.x>wall[1])
    pos.x=wall[1];
  if(pos.y<wall[2])
    pos.y=wall[2];
  if(pos.y>wall[3])
    pos.y=wall[3];
  if(pos.z<wall[4])
    pos.z=wall[4];
  if(pos.z>wall[5])
    pos.z=wall[5];
}



__global__ void initIsend()
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    isend[num] = 2;
  }
  __syncthreads();
}

__global__ void flockUpdate()
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    float3 tmp;
    float3 tmpv;
    tmp = getPos(num);
    tmpv = getDir(num);
//    wallCheck(num,tmp,tmpv);
    __syncthreads();
    Avg avg;
    avg.Rpos = make_float3(0,0,0);
    avg.rpos = make_float3(0,0,0);
    avg.Rvel = make_float3(0,0,0);
    avg.rvel = make_float3(0,0,0);
    avg.countR = 0;
    avg.countr = 0;    
    calculateAvg(num,avg);
    if(avg.countR>0){
       avg.Rpos = avg.Rpos/avg.countR;
       avg.Rvel = avg.Rvel/avg.countR;
    }
    if(avg.countr>0){
       avg.rpos = avg.rpos;///avg.countr;
    //   avg.rvel = avg.rvel/avg.countr;
    }
    // -----------------------------
    // please update position here
    //------------------------------
    // avg.Rpos average position within R
    // avg.rpos pos/r  position within r
    // avg.Rvel average velocity within R
    // avg.rvel average velocity within r    
    // above variable are float3.
    // wall[0~5]
    tmpv=normalize(tmpv);
    if (avg.countR>0)
    {
//	tmpv=normalize(avg.Rpos-tmp);
        tmpv = normalize(tmpv + normalize(avg.Rpos - tmp)*para.C);
        tmpv = normalize(tmpv + normalize(avg.Rvel)*para.A);    
     
  
//	tmpv=normalize(tmpv+normalize(avg.Rpos-tmp));
  //    tmpv = normalize(tmpv + normalize(avg.Rvel));    
    
    if (avg.countr>0){
 //     tmpv = normalize(avg.rpos)*para.S;
     // tmpv
     tmpv =/* normalize(tmpv + */normalize(avg.rpos)*para.S;
      //tmpv = tmpv+normalize(make_float3(1,0,0))*para.S;
      //tmpv = normalize(tmpv);
	
    }

    }
    wallCheck(num,tmp,tmpv);
    tmp = tmp+(tmpv*para.dt);
    setPos(num,tmp);
    setDir(num,tmpv); 
  }
  
  __syncthreads();
}

void FlockSim::update()
{
  dim3 Grid(Grid_Dim_x, Grid_Dim_y);		//Grid structure
  dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device
  initIsend<<<Grid,Block>>>();  
  flockUpdate<<<Grid,Block>>>();
  convertDir<<<Grid,Block>>>(); // convert xyz velocity to angle
}
