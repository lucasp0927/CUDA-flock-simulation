#include "hip/hip_runtime.h"
#include "flocksim.h"

FlockSim::FlockSim(int size, int thread_n,WorldGeo& wg):_size(size),_thread_n(thread_n),_wg(wg)
{
  _thread_handles = (pthread_t*)malloc(_thread_n*sizeof(pthread_t));
  cout << "Using " << _thread_n << " threads!" << endl;
  _kt = new KdTree(_thread_n,size,&_wg);
  cout << "random initialize"<<endl;
  _kt->randInit();
  cout << "allocate memory on GPU"<< endl;
  _psize = Node::getPSize();
  _pos = Node::getPos();
  _xyz_dir = Node::getDir();  
  _tree = Node::getTree();  
  hipMalloc((void**)&_dev_pos,_size*_psize*sizeof(float));
  hipMalloc((void**)&_dev_tree,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_xyz_dir,_size*3*sizeof(int));
  hipMalloc((void**)&_dev_ang_dir,_size*3*sizeof(int));  
  _ang_dir = new float[_size*3*sizeof(float)];
  // cuda grid sructure
  Block_Dim_x = 512;
  Block_Dim_y = 1;  
  Grid_Dim_x = (int)_size/Block_Dim_x +1;
  if (Grid_Dim_x > 65565)
    cerr << "too many block!" << endl;
  Grid_Dim_y = 1;
}

FlockSim::~FlockSim()
{
  hipFree(&_dev_pos);         // will render need this mem?
  hipFree(&_dev_tree);
  hipFree(&_dev_xyz_dir);
  hipFree(&_dev_ang_dir);  
  delete [] _ang_dir;
}

void FlockSim::initializeGpuData()
{
  hipMemcpy(_dev_pos, _pos, _size*_psize*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(_dev_xyz_dir, _xyz_dir, _size*3*sizeof(float),hipMemcpyHostToDevice);    
}

void FlockSim::cpytree2dev()
{
  hipMemcpy(_dev_tree, _tree, _size*3*sizeof(int),hipMemcpyHostToDevice);      
}

void FlockSim::cpy2host()
{
  hipMemcpy(_pos, _dev_pos, _size*_psize*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(_ang_dir, _dev_ang_dir, _size*3*sizeof(float),hipMemcpyDeviceToHost);  
}

void FlockSim::makeTree()
{
  _kt->findRoot();  
  _kt->construct();
  ConstructTree(_thread_n,_kt,_thread_handles);
  if(_kt->checkTree())
    cout << "correct" << endl;
}
__global__  void convertDir(float* _xyz_dir,float* _ang_dir,int size)
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    float r = 0.0;
    for (int i = 0; i < 3; ++i)
    {
      r += _xyz_dir[num*3+i]*_xyz_dir[num*3+i];
    }
    r = sqrt(r);
    _ang_dir[num*3] = -180.0*acos(_xyz_dir[num*3]/r)/M_PI;    
    _ang_dir[num*3+1] = _xyz_dir[num*3+2];
    _ang_dir[num*3+2] = -1.0*_xyz_dir[num*3+1];    
  }
}

__device__ void calculate(int root,int num,float R,float r,float3* avgRpos,float3* rpos,float3* avgRvel, float3* avgrvel)
{
  int cur = root;
  cur = goDown(cur,num,dis);
  while (cur != root)
  {
    if (move(cur,num,dis))
      cur = goDown(cur,num,dis);
  }
}

__global__ void flockUpdate(float* _pos,float* _xyz_dir,int size)
{
  int num = threadIdx.x + blockDim.x * blockIdx.x;
  if (num < size)
  {
    float3 avgRpos,rpos,avgRvel,avgrvel;    
    calculate(num,R,r,&avgRpoa,&rpos,&avgRvel,&avgrvel);
    
  }
}


void FlockSim::update()
{
    dim3 Grid(Grid_Dim_x, Grid_Dim_y);		//Grid structure
    dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device

    
    convertDir<<<Grid,Block>>>(_dev_xyz_dir,_dev_ang_dir,_size); // convert xyz velocity to angle
}
