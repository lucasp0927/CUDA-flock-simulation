#include "flocksim.h"

FlockSim::FlockSim(int size, int thread_n,WorldGeo& wg):_size(size),_thread_n(thread_n),_wg(wg)
{
  _thread_handles = (pthread_t*)malloc(_thread_n*sizeof(pthread_t));
  cout << "Using " << _thread_n << " threads!" << endl;
  _kt = new KdTree(_thread_n,size,&_wg);
  cout << "random initialize"<<endl;
  _kt->randInit();
  cout << "allocate memory on GPU"<< endl;
  _psize = Node::getPSize();
  _data = Node::getData();
  _tree = Node::getTree();  
  hipMalloc((void**)&_dev_data,_size*_psize*sizeof(float));
  hipMalloc((void**)&_dev_tree,_size*3*sizeof(int));    
}

FlockSim::~FlockSim()
{
  hipFree(&_dev_data);         // will render need this mem?
  hipFree(&_dev_tree);  
}

void FlockSim::cpy2dev()
{
  hipMemcpy(_dev_data, _data, _size*_psize*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(_dev_tree, _tree, _size*3*sizeof(int),hipMemcpyHostToDevice);      
}

void FlockSim::cpy2host()
{
  hipMemcpy(_data, _dev_data, _size*_psize*sizeof(float),hipMemcpyDeviceToHost);
}

void FlockSim::makeTree()
{
  _kt->findRoot();  
  _kt->construct();
  ConstructTree(_thread_n,_kt,_thread_handles);
}

void FlockSim::update()
{
  
}
