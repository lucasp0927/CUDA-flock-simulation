#include "hip/hip_runtime.h"
#include "node.h"
void normalize(float* f,int size)
{
  float len = 0.0;
  for (int i = 0; i < size; ++i)
    len += (*(f+i))*(*(f+i));
  len = sqrt(len);
  for (int i = 0; i < size; ++i)
    *(f+i) /= len;
}

float randRange(float a,float b)
{
  // give a random float between a and b
  return a+(b-a)*((float)rand()/RAND_MAX);
}

bool Node:: _static_init = false;
int Node::_dim = 0;
int Node::_size = 0;
int Node::_psize = 0;
float* Node::_pos = NULL;
int* Node::_depth = NULL;
float* Node::_xyz_dir = NULL;
int* Node::_tree = NULL;

Node::Node()
{
}

Node::~Node()
{
  clear();
  if (_static_init)
  {
    delete [] _pos;
    _pos = NULL;
    delete [] _tree;
    _tree = NULL;
    delete [] _xyz_dir;
    _xyz_dir = NULL;
    delete [] _depth;
    _depth = NULL;
    _static_init = false;
  }

}

void Node::init(int dim,int idx,int size)
{
  _idx = idx;
  _list = NULL;
  _llist = NULL;
  _rlist = NULL;
  if (!_static_init)
  {
    assert(dim == 2||dim == 3);
    if (dim == 3)
      _psize = 3;
    else if (dim == 2)
      _psize = 2;
    _size = size;
    _dim = dim;
    _tree = new int[3*_size];
    _xyz_dir = new float[_dim*_size];
    _depth = new int[_size];        
    _pos = new float[_psize*_size];
    _static_init = true;
  }
  
}
void Node::setIdx(int i){  _idx = i;}
inline void Node::setParent(int p){  _tree[3*_idx] = p;}
inline void Node::setLChild(int l){  _tree[3*_idx+1] = l;}
inline void Node::setRChild(int r){  _tree[3*_idx+2] = r;}
int Node::getParent() const{  return _tree[3*_idx];}
int Node::getLChild() const{  return _tree[3*_idx+1];}
int Node::getRChild() const{  return _tree[3*_idx+2];}
int Node::getIdx() const{return _idx;}
vector<int>* Node::getList() const{return _list;}
vector<int>* Node::getLList() const{return _llist;}
vector<int>* Node::getRList() const{return _rlist;}
bool Node::isEnd()
{
  if (getLChild() == _idx && getRChild() == _idx)
    return true;
  else return false;
}

void Node::buildRootList(int size)
{
  setParent(_idx);
  if (_list == NULL)
    _list = new vector<int>;
  else
    _list->clear();
  //_list->reserve(size);    
  for (int i = 0; i < size; ++i)
    (*_list).push_back(i);
  setDepth(_idx,0);
}

void Node::separateList()
{
  int dim = getDepth(_idx)%_dim;
  if (_rlist == NULL)
    _rlist = new vector<int>;
  else _rlist->clear();
  
  if (_llist == NULL)
    _llist = new vector<int>;
  else _llist->clear();  
  // _rlist->reserve(_list->size()/2);
  // _llist->reserve(_list->size()/2);
  for(vector<int>::iterator it = _list->begin(); it != _list->end(); ++it) {
    if (*it != _idx)
    {
      assert(*it >= 0 && *it < _size);
      assert(_idx >= 0 && _idx < _size);      
      if (getPos(*it,dim) > getPos(_idx,dim))
        _rlist->push_back(*it);
      else
        _llist->push_back(*it);
    }
  }
  _list->clear();
}


void Node::setPos(int dim,float pos)
{
  _pos[_idx*_psize+dim] = pos;  
}

void Node::setDir(int dim,float dir)
{
  _xyz_dir[_idx*_psize+dim] = dir;  
}


bool Node::Less::operator() (const int & a, const int& b)
{
  assert(a >= 0&& a < myNode->_size);
  assert(b >= 0&& b < myNode->_size);
  return (Node::getPos(a,Node::getDepth(myNode->getIdx())%Node::getDim()) < Node::getPos(b,Node::getDepth(myNode->getIdx())%Node::getDim()));
} 


void quick_sort (int *a, int n,int depth) {
  int dim = Node::getDim();
    if (n < 2)
        return;
    float p = Node::getPos(a[n / 2],depth%dim);
    int *l = a;
    int *r = a + n - 1;
    while (l <= r) {
      while (Node::getPos(*l,depth%dim) < p)
            l++;
      while ( Node::getPos(*r,depth%dim)> p)
            r--;
        if (l <= r) {
            int t = *l;
            *l++ = *r;
            *r-- = t;
        }
    }
    quick_sort(a, r - a + 1, depth);
    quick_sort(l, a + n - l, depth);
}

int Node::median(int sample_sz,vector<int>* list,bool next,struct drand48_data *buffer)
{
  double randnum;
  //  vector<int> sample;
  vector<int>* sample;
  sample = new vector<int>;
  int tmp;
  if (list == NULL)
  {
    sample_sz = _size < sample_sz? _size:sample_sz;
    assert (sample_sz == SAMPLESIZE || sample_sz == _size );
    sample->clear();
    int count = 0;
    while (count < sample_sz)
    {
      if (buffer == NULL)
        tmp = rand() % (_size);
      else
      {
        drand48_r(buffer, &randnum);
        tmp =  (int)randnum*(_size);
      }
      //    if (find(sample.begin(),sample.end(),tmp) == sample.end())
      // {
      if (tmp >= _size)
        tmp = _size-1;
      assert(tmp >=0 && tmp < _size);
      sample->push_back(tmp);
      count++;
      //      }
    }
    sort(sample->begin(),sample->end(),Less(this));        
    //quick_sort (sample, sample_sz,getDepth(_idx));
    
  }
  else
  {
    if(next)
      _depth[_idx]++;
    assert(list->size() != 0);
    sample_sz = list->size() < sample_sz? list->size():sample_sz;
    assert (sample_sz == SAMPLESIZE || sample_sz == list->size());    
    sample->clear();
    //    sample.reserve(sample_sz);      
    int count = 0;
    while (count < sample_sz)
    {
      if (buffer == NULL)
        tmp = rand() % (list->size());        
      else
      {
        drand48_r(buffer, &randnum);
        tmp =  (int)randnum*(list->size());
      }      

      //      if (find(sample.begin(),sample.end(),tmp) == sample.end())
      //      {
      if (tmp >= list->size() )
        tmp = list->size()-1;
      sample->push_back(tmp);
      count++;
      //      }
    };
    for (int i = 0; i < sample_sz; ++i)
    {
      int tmp;
      tmp = (*list)[(*sample)[i]];
      assert(tmp >= 0 && tmp < _size);
      (*sample)[i] = tmp;
    }
    
    sort(sample->begin(),sample->end(),Less(this));    
//quick_sort (sample, sample_sz,getDepth(_idx));          
    if(next)
      _depth[_idx]--;
  }
  
  int result = (*sample)[sample_sz/2];    
  delete sample;
  return result;  
}

void Node::setList(vector<int>* list)
{
  if (_list != NULL)
  {
    delete _list;
    _list = NULL;
  }
  _list = list;
}

void Node::setChild(Node* left,Node* right)
{
  if (left != NULL)
  {
    setLChild(left->getIdx());
    left->setParent(_idx);
    Node::setDepth(left->getIdx(),_depth[_idx]+1);
    left->setList(_llist);
    _llist = NULL;
  }
  else
    setLChild(_idx);
  
  if (right != NULL)
  {
    setRChild(right->getIdx());
    right->setParent(_idx);  
    Node::setDepth(right->getIdx(),Node::getDepth(_idx)+1);
    right->setList(_rlist);
    _rlist = NULL;
  }
  else
    setRChild(_idx);
}

int Node::leftmedian(struct drand48_data *buffer)
{
  return median(100,_llist,true,buffer);
}

int Node::rightmedian(struct drand48_data *buffer)
{
  return median(100,_rlist,true,buffer);  
}

float Node::distance(int idx)
{
  float d = 0.0;
  float tmp;
  for (int i = 0; i < _dim; ++i)
  {
    tmp = getPos(idx,i) - getPos(_idx,i);
    d += tmp*tmp;
  }
  d = sqrt(d);
  return d;
}

void Node::clear()
{
  if (_list != NULL)
  {
    delete _list;
    _list = NULL;
  }

  if (_llist != NULL)
  {
    delete _llist;
    _llist = NULL;
  }
  
  if (_rlist != NULL)
  {
    delete _rlist;
    _rlist = NULL;
  }
}

  
ostream &operator <<(ostream &os,Node& n)
{
  os << setw(3) << n.getIdx();
  for (int i = 0; i < Node::getDim(); ++i)
  {
    os << setw(10) << n.getPos(n.getIdx(),i);
  }
  os << setw(5) << "p:" << setw(5) << n.getParent();
  os << setw(5) << "l:" << setw(5) << n.getLChild();
  os << setw(5) << "r:" << setw(5) << n.getRChild();
  os << setw(5) << "d:" << setw(5) << Node::getDepth(n.getIdx());
  os << setw(5) << "e:" << setw(5) << n.isEnd();    
  return os;
}

/*
  angle:  -180*acos( dx / sqrt (  dx*dx + dy*dy  + dz *dz ))/M_PI
  rX:0
  rY: dz
  rZ: -dy
  dx dy dz是方向向
*/
