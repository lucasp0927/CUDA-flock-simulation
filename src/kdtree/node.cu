#include "hip/hip_runtime.h"
#include "node.h"
void normalize(float* f,int size)
{
  float len = 0.0;
  for (int i = 0; i < size; ++i)
    len += (*(f+i))*(*(f+i));
  len = sqrt(len);
  for (int i = 0; i < size; ++i)
    *(f+i) /= len;
}

float randRange(float a,float b)
{
  // give a random float between a and b
  return a+(b-a)*((float)rand()/RAND_MAX);
}

bool Node:: _static_init = false;
int Node::_dim = 0;
int Node::_size = 0;
int Node::_psize = 0;
float* Node::_data = NULL;
int* Node::_tree = NULL;

Node::Node()
{
}

Node::~Node()
{
  if (_static_init)
  {
    delete [] _data;
    delete [] _tree;
    _static_init = false;
  }
}

void Node::init(int dim,int idx,int size)
{
  _idx = idx;
  _depth = 0;
  _list = NULL;
  _llist = NULL;
  _rlist = NULL;
  if (!_static_init)
  {
    assert(dim == 2||dim == 3);
    if (dim == 3)
      _psize = 8;
    else if (dim == 2)
      _psize = 4;
    _size = size;
    _dim = dim;
    _tree = new int[3*_size];
    _data = new float[_psize*_size];
    _static_init = true;
  }
  
}
void Node::setIdx(int i){  _idx = i;}
inline void Node::setParent(int p){  _tree[3*_idx] = p;}
inline void Node::setLChild(int l){  _tree[3*_idx+1] = l;}
inline void Node::setRChild(int r){  _tree[3*_idx+2] = r;}
int Node::getParent() const{  return _tree[3*_idx];}
int Node::getLChild() const{  return _tree[3*_idx+1];}
int Node::getRChild() const{  return _tree[3*_idx+2];}
void Node::setDepth(int d){  _depth = d;}
int Node::getDepth() const{return _depth;}
int Node::getDim() const{return _dim;}
int Node::getIdx() const{return _idx;}
vector<int>* Node::getList() const{return _list;}
vector<int>* Node::getLList() const{return _llist;}
vector<int>* Node::getRList() const{return _rlist;}
bool Node::isEnd()
{
  if (getLChild() == _idx && getRChild() == _idx)
    return true;
  else return false;
}

void Node::buildRootList(int size)
{
  setParent(_idx);
  if (_list == NULL)
    _list = new vector<int>;
  else
    _list->clear();
  
  _list->reserve(size);    
  for (int i = 0; i < size; ++i)
    (*_list).push_back(i);
  _depth = 0;  
}

void Node::separateList()
{
  int dim = _depth%_dim;
  if (_rlist == NULL)
    _rlist = new vector<int>;
  if (_llist == NULL)
    _llist = new vector<int>;
  for(vector<int>::iterator it = _list->begin(); it != _list->end(); ++it) {
    if (*it != _idx)
    {
      if (getPos(*it,dim) > getPos(_idx,dim))
        _rlist->push_back(*it);
      else
        _llist->push_back(*it);
    }
  }
  _list->clear();
}
float Node::getPos(int idx,int dim) const
{
   assert (dim < _dim);
   return _data[idx*_psize+dim];   
}
float Node::getDir(int dim) const
{
  assert (dim < _dim);
  return _data[_idx*_psize+_dim+dim];
}
void Node::setPos(int dim,float pos)
{
  assert (dim < _dim);
  _data[_idx*_psize+dim] = pos;  
}
void Node::setDir(float* dir)
{
  normalize(dir,_dim);
  for (int i = 0; i < _dim; ++i)
    _data[_idx*_psize+_dim+i] = dir[i];      
}
bool Node::Less::operator() (const int & a, const int& b)
{
      return (myNode->getPos(a,myNode->getDepth()%myNode->getDim()) < myNode->getPos(b,myNode->getDepth()%myNode->getDim()));          
} 

int Node::median(int sample_sz,vector<int>* list,bool next,struct drand48_data *buffer)
{
  double randnum;
  vector<int> sample;
  if (list == NULL)
  {
    //    assert(_depth == 0);
    sample_sz = _size < sample_sz? _size:sample_sz;
    sample.reserve(sample_sz);  
    // test all points
    int count = 0;
    int tmp;
    while (count != sample_sz)
    {
      if (buffer == NULL)
        tmp = rand() % _size;
      else
      {
        drand48_r(buffer, &randnum);
        tmp =  tmp*(_size);
      }
        //    if (find(sample.begin(),sample.end(),tmp) == sample.end())
        // {
        sample.push_back(tmp);
        count++;
        //      }
    };
    sort(sample.begin(),sample.end(),Less(this));
    return sample[sample.size()/2];    
  }
  else
  {
    if(next)
      _depth++;
    assert(list->size() != 0);
    sample_sz = list->size() < sample_sz? list->size():sample_sz;
    sample.reserve(sample_sz);      
    int count = 0;
    int tmp;
    while (count != sample_sz)
    {
      if (buffer == NULL)
        tmp = rand() % (list->size());        
      else
      {
        drand48_r(buffer, &randnum);
        tmp =  randnum*(list->size());
      }      

      //      if (find(sample.begin(),sample.end(),tmp) == sample.end())
      //      {
        sample.push_back(tmp);
        count++;
        //      }
    };
    for (int i = 0; i < sample.size(); ++i)
      sample[i] = (*list)[sample[i]];
    sort(sample.begin(),sample.end(),Less(this));
    if(next)
      _depth--;
    return sample[sample.size()/2];
  }
}

void Node::setList(vector<int>* list)
{
  if (_list != NULL)
  {
    delete _list;
    _list = NULL;
  }
  _list = list;
}

void Node::setChild(Node* left,Node* right)
{
  if (left != NULL)
  {
  setLChild(left->getIdx());
  left->setParent(_idx);
  left->setDepth(_depth+1);
  left->setList(_llist);
  _llist = NULL;
  }
  else
    setLChild(_idx);
  
  if (right != NULL)
  {
    setRChild(right->getIdx());
    right->setParent(_idx);  
    right->setDepth(_depth+1);
    right->setList(_rlist);
    _rlist = NULL;
  }
  else
    setRChild(_idx);
}

int Node::leftmedian(struct drand48_data *buffer)
{
  return median(100,_llist,true,buffer);
}

int Node::rightmedian(struct drand48_data *buffer)
{
  return median(100,_rlist,true,buffer);  
}

float Node::distance(int idx)
{
  float d = 0.0;
  float tmp;
  for (int i = 0; i < _dim; ++i)
  {
    tmp = getPos(idx,i) - getPos(_idx,i);
    d += tmp*tmp;
  }
  d = sqrt(d);
  return d;
}

void Node::clear()
{
  if (_list != NULL)
  {
    delete _list;
    _list = NULL;
  }

  if (_llist != NULL)
  {
    delete _llist;
    _llist = NULL;
  }
  
  if (_rlist != NULL)
  {
    delete _rlist;
    _rlist = NULL;
  }
}

  
ostream &operator <<(ostream &os,Node& n)
{
  os << setw(3) << n.getIdx();
  for (int i = 0; i < n.getDim(); ++i)
  {
    os << setw(10) << n.getPos(n.getIdx(),i);
  }
  os << setw(5) << "p:" << setw(5) << n.getParent();
  os << setw(5) << "l:" << setw(5) << n.getLChild();
  os << setw(5) << "r:" << setw(5) << n.getRChild();
  os << setw(5) << "d:" << setw(5) << n.getDepth();
  os << setw(5) << "e:" << setw(5) << n.isEnd();    
  return os;
}

/*
  angle:  -180*acos( dx / sqrt (  dx*dx + dy*dy  + dz *dz ))/M_PI
  rX:0
  rY: dz
  rZ: -dy
  dx dy dz是方向向
*/
