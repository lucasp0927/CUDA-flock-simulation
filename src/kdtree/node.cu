#include "hip/hip_runtime.h"
#include "node.h"
void normalize(float* f,int size)
{
  float len = 0.0;
  for (int i = 0; i < size; ++i)
    len += (*(f+i))*(*(f+i));
  len = sqrt(len);
  for (int i = 0; i < size; ++i)
    *(f+i) /= len;
}

float randRange(float a,float b)
{
  // give a random float between a and b
  return a+(b-a)*((float)rand()/RAND_MAX);
}

bool Node:: _static_init = false;
int Node::_dim = 0;
int Node::_size = 0;
int Node::_psize = 0;
float* Node::_pos = NULL;
int* Node::_depth = NULL;
float* Node::_xyz_dir = NULL;
int* Node::_tree = NULL;

Node::Node()
{
}

Node::~Node()
{
  clear();
  if (_static_init)
  {
    //    delete [] _pos;
    free(_pos);
    _pos = NULL;
    delete [] _tree;
    _tree = NULL;
    delete [] _xyz_dir;
    _xyz_dir = NULL;
    delete [] _depth;
    _depth = NULL;
    _static_init = false;
  }

}

void Node::init(int dim,int idx,int size)
{
  _idx = idx;
  _list = NULL;
  _llist = NULL;
  _rlist = NULL;
  if (!_static_init)
  {
    assert(dim == 2||dim == 3);
    if (dim == 3)
      _psize = 3;
    else if (dim == 2)
      _psize = 2;
    _size = size;
    _dim = dim;
    _tree = new int[3*_size];
    _xyz_dir = new float[_dim*_size];
    _depth = new int[_size];        
    _pos =(float*) malloc(_psize*_size*sizeof(float));
    //new float[_psize*_size];
    _static_init = true;
  }
  
}
void Node::setIdx(int i){  _idx = i;}
inline void Node::setParent(int p){  _tree[3*_idx] = p;}
inline void Node::setLChild(int l){  _tree[3*_idx+1] = l;}
inline void Node::setRChild(int r){  _tree[3*_idx+2] = r;}
int Node::getParent() const{  return _tree[3*_idx];}
int Node::getLChild() const{  return _tree[3*_idx+1];}
int Node::getRChild() const{  return _tree[3*_idx+2];}
int Node::getIdx() const{return _idx;}
vector<int>* Node::getList() const{return _list;}
vector<int>* Node::getLList() const{return _llist;}
vector<int>* Node::getRList() const{return _rlist;}
bool Node::isEnd()
{
  if (getLChild() == _idx && getRChild() == _idx)
    return true;
  else return false;
}

void Node::buildRootList(int size)
{
  setParent(_idx);
  if (_list == NULL)
    _list = new vector<int>;
  else
    _list->clear();
  //_list->reserve(size);    
  for (int i = 0; i < size; ++i)
    (*_list).push_back(i);
  setDepth(_idx,0);
}

void Node::separateList()
{
  int dim = getDepth(_idx)%_dim;
  if (_rlist == NULL)
    _rlist = new vector<int>;
  else _rlist->clear();
  
  if (_llist == NULL)
    _llist = new vector<int>;
  else _llist->clear();  
  // _rlist->reserve(_list->size()/2);
  // _llist->reserve(_list->size()/2);
  assert(dim >= 0 && dim < 3);
  float piv = getPos(_idx,dim);
  for(vector<int>::iterator it = _list->begin(); it != _list->end(); ++it) {
    if (*it != _idx)
    {
      assert(*it >= 0 && *it < _size);
      assert(_idx >= 0 && _idx < _size);
      assert(dim >= 0 && dim < 3);
      if (getPos(*it,dim) > piv)
        _rlist->push_back(*it);
      else
        _llist->push_back(*it);
    }
  }
  _list->clear();
  delete _list;
  _list = NULL;
}

void Node::setPos(int dim,float pos)
{
  _pos[_idx*_psize+dim] = pos;  
}

void Node::setDir(int dim,float dir)
{
  _xyz_dir[_idx*_psize+dim] = dir;  
}

int compare (const void* a,const void* b)
{
  return(((tuplet*)a)->pos - ((tuplet*)b)->pos);
}

int Node::median(int sample_sz,vector<int>* list,bool next,struct drand48_data *buffer)
{
  double randnum;
  //  vector<int> sample;
  tuplet* sample;    
  //  sample = new vector<int>;
  int tmp;

  //  cerr << list->size();
  if (list == NULL)
  {
    int ax = getDepth(_idx)%getDim();    
    sample_sz = _size < sample_sz? _size:sample_sz;
    sample = (tuplet*) malloc(sample_sz*sizeof(tuplet));
    assert (sample_sz == SAMPLESIZE || sample_sz == _size );
    for (int i = 0; i < sample_sz; ++i)
    {
      if (buffer == NULL)
        tmp = rand() % (_size);
      else
      {
        drand48_r(buffer, &randnum);
        tmp =  (int)(randnum*_size);
      }
      assert(tmp >=0 && tmp < _size);
      // if (tmp >= _size)
      //   tmp = _size-1;
      assert(tmp >=0 && tmp < _size);
      //      sample.push_back(tuplet(tmp,Node::getPos(tmp,ax)));
      sample[i].idx = tmp;
      sample[i].pos = Node::getPos(tmp,ax);
    }
    //sort(sample.begin(),sample.end(),compare);        
    qsort(sample,sample_sz,sizeof(tuplet),compare);
  }
  else
  {
    if(next)
      _depth[_idx]++;
    int ax = getDepth(_idx)%getDim();        
    assert(list->size() != 0);
    sample_sz = list->size() < sample_sz? list->size():sample_sz;
    sample = (tuplet*) malloc(sample_sz*sizeof(tuplet));
    assert (sample_sz == SAMPLESIZE || sample_sz == list->size());    
    //    sample.clear();
    //    sample.reserve(sample_sz);
    for (int i = 0; i < sample_sz; ++i)
    {
      if (buffer == NULL)
        tmp = rand() % (list->size());        
      else
      {
        drand48_r(buffer, &randnum);
        tmp =  (int)(randnum*list->size());
      }
      assert(tmp >= 0 && tmp < list->size());
      // if (tmp >= list->size() )
      //   tmp = list->size()-1;
      tmp = (*list)[tmp];
      assert(tmp >= 0 && tmp < _size);
      sample[i].idx = tmp;
      sample[i].pos = Node::getPos(tmp,ax);      
      //      sample.push_back(tuplet(tmp,Node::getPos(tmp,ax)));      
    }
    // vector<int>::iterator it;
    // cout << "sample contains:";
    // for (it=sample.begin(); it!=sample.end(); ++it)
    //   cout << " " << *it;
    // cout << endl;
    assert(getDepth(_idx) >= 0);    
    assert(ax >= 0 && ax < 3);    
    //    sort(sample.begin(),sample.end(),compare);    
//quick_sort (sample, sample_sz,getDepth(_idx));
    qsort(sample,sample_sz,sizeof(tuplet),compare);
    if(next)
      _depth[_idx]--;
  }
  
  int result = sample[sample_sz/2].idx;
  free(sample);
  //  delete sample;
  return result;  
}

void Node::setList(vector<int>* list)
{
  if (_list != NULL)
  {
    delete _list;
    _list = NULL;
  }
  _list = list;
}

void Node::setChild(Node* left,Node* right)
{
  if (left != NULL)
  {
    int lidx = left->getIdx();
    setLChild(lidx);
    left->setParent(_idx);
    Node::setDepth(lidx,_depth[_idx]+1);
    left->setList(_llist);
    _llist = NULL;
  }
  else
  {
    setLChild(_idx);
    delete _llist;
    _llist = NULL;
  }
    
  if (right != NULL)
  {
    int ridx = right->getIdx();
    setRChild(ridx);
    right->setParent(_idx);  
    Node::setDepth(ridx,Node::getDepth(_idx)+1);
    right->setList(_rlist);
    _rlist = NULL;
  }
  else
  {
    setRChild(_idx);
    delete _rlist;
    _rlist = NULL;
  }
  
}

int Node::leftmedian(struct drand48_data *buffer)
{
  return median(100,_llist,true,buffer);
}

int Node::rightmedian(struct drand48_data *buffer)
{
  return median(100,_rlist,true,buffer);  
}

float Node::distance(int idx)
{
  float d = 0.0;
  float tmp;
  for (int i = 0; i < _dim; ++i)
  {
    tmp = getPos(idx,i%3 ) - getPos(_idx,i%3 );
    d += tmp*tmp;
  }
  d = sqrt(d);
  return d;
}

void Node::clear()
{
  if (_list != NULL)
  {
    delete _list;
    _list = NULL;
  }

  if (_llist != NULL)
  {
    delete _llist;
    _llist = NULL;
  }
  
  if (_rlist != NULL)
  {
    delete _rlist;
    _rlist = NULL;
  }
}

  
ostream &operator <<(ostream &os,Node& n)
{
  os << setw(3) << n.getIdx();
  for (int i = 0; i < Node::getDim(); ++i)
  {
    os << setw(10) << n.getPos(n.getIdx(),i%3 );
  }
  os << setw(5) << "p:" << setw(5) << n.getParent();
  os << setw(5) << "l:" << setw(5) << n.getLChild();
  os << setw(5) << "r:" << setw(5) << n.getRChild();
  os << setw(5) << "d:" << setw(5) << Node::getDepth(n.getIdx());
  os << setw(5) << "e:" << setw(5) << n.isEnd();    
  return os;
}

/*
  angle:  -180*acos( dx / sqrt (  dx*dx + dy*dy  + dz *dz ))/M_PI
  rX:0
  rY: dz
  rZ: -dy
  dx dy dz是方向向
*/
