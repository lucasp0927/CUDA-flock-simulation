#include <stdio.h>
#include<stdlib.h>
#include<iostream>
#include"FlockSim.h"

#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>

#define W 640
#define H 480

using namespace std;

FlockSim* Fsim;

void idle(){
  glutPostRedisplay();
}


void display() {
  printf("fuckyou]\n");
  glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
  glLoadIdentity();
  glBegin(GL_POINTS);
  glColor3f(1.0f,1.0f,1.0f);
  for(int i=0;i<Fsim->F.size;i++){
    printf("%d\n",i);
    if(1){      printf("i=%d x=%f y=%f\n",i,Fsim->F.flock[i].x,Fsim->F.flock[i].y);}
    glVertex2f(2*Fsim->F.flock[i].x/W,2*Fsim->F.flock[i].y/H);
    //bird[i].move();
    //bird[i].turns(0.5);
  }
  Fsim->update_flock(1.0);
  Fsim->copy2host();
  glEnd();
  glFinish();
}




int main(int argc, char **argv)
{
  int flock_size=atoi(argv[1]);
  int wall_size=atoi(argv[2]);  //wall_size is a data menber of FlockSim
  cout << "Simulate a flock with " << flock_size << " agents." << endl;
  FlockSim Fs = FlockSim(flock_size,wall_size);
  Fsim = &Fs;
  /*
  Fsim->printFlock();
  Fsim->update_flock(100.0);     // big dt for testing
  Fsim->copy2host();
  Fsim->printFlock();
  Fsim->update_flock(100.0);     // big dt for testing
  Fsim->copy2host();
  Fsim->printFlock();
  */

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_RGBA);
  glutInitWindowSize(W, H);
  glutCreateWindow("flock");
  glutDisplayFunc(display);
  glutIdleFunc(idle);
  glutMainLoop();

  return 0;
}
