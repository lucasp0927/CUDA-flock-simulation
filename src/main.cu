#include <stdio.h>
#include<stdlib.h>
#include<iostream>
#include"FlockSim.h"

#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>

#define W 640
#define H 480

using namespace std;

 int first;
int flock_size;
int wall_size;
FlockSim Fsim;
void idle(){
    glutPostRedisplay();
}


void display() {
    printf("fuckyou]\n");
    glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
    glLoadIdentity();
    glBegin(GL_POINTS);
    glColor3f(1.0f,1.0f,1.0f);
    for(int i=0;i<flock_size+1;i++){
      printf("%d\n",i);
  if(1){      printf("i=%d x=%f y=%f\n",i,Fsim.F.flock[i].x,Fsim.F.flock[i].y);}
        glVertex2f(2*Fsim.F.flock[i].x/W,2*Fsim.F.flock[i].y/H);
        //bird[i].move();
        //bird[i].turns(0.5);
    }
    if(first==1){
       Fsim.printFlock();
       first=0;  
    }
    Fsim.update_flock(100.0);
    Fsim.printFlock();
    glEnd();
    glFinish();
}




int main(int argc, char **argv)
{
  first=1;
  flock_size=atoi(argv[1]);
  wall_size=atoi(argv[2]);  
  cout << "Simulate a flock with " << flock_size << " agents." << endl;
  Fsim=FlockSim(flock_size,wall_size);
  //Fsim.printFlock();
      //  printf("x=%f y=%f\n",Fsim.F.flock[0].x,Fsim.F.flock[0].y);
//printf("fuck\n");
 // Fsim.update_flock(100.0);     // big dt for testing
  //Fsim.printFlock();
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA);
    glutInitWindowSize(W, H);
    glutCreateWindow("flock");
    glutDisplayFunc(display);
    glutIdleFunc(idle);
    glutMainLoop();

  return 0;
}
