#include<stdlib.h>
#include<iostream>
#include"FlockSim.h"

#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>

using namespace std;

int main(int argc, char *argv[])
{
  int flock_size = atoi(argv[1]);
  int wall_size = atoi(argv[2]);  
  cout << "Simulate a flock with " << flock_size << " agents." << endl;
  FlockSim Fsim(flock_size,wall_size);
  Fsim.printFlock();
  
  return 0;
}
